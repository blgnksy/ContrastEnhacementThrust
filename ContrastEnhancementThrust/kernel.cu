#include "hip/hip_runtime.h"

#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <npp.h>
#include <stdio.h>


//https://github.com/thrust/thrust/wiki/Quick-Start-Guide
// Function Protypes.
thrust::host_vector<Npp8u>
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray);

void
WritePGM(char * sFileName, Npp8u * pDst_Host, int nWidth, int nHeight, int nMaxGray);

int main()
{
	thrust::host_vector<Npp8u> pSrc_Host;
	int   nWidth, nHeight, nMaxGray, nNormalizer;

	std::cout << "THRUST VERSION" << std::endl;

	// Load image to the host.
	std::cout << "Load PGM file." << std::endl;
	pSrc_Host = LoadPGM("lena_before.pgm", nWidth, nHeight, nMaxGray);
	//pDst_Host = new Npp8u[nWidth * nHeight];
	
    return 0;
}

// Load PGM file.
thrust::host_vector<Npp8u>
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray)
{
	char aLine[256];
	FILE * fInput = fopen(sFileName, "r");
	if (fInput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	// First line: version
	fgets(aLine, 256, fInput);
	std::cout << "\tVersion: " << aLine;
	// Second line: comment
	fgets(aLine, 256, fInput);
	std::cout << "\tComment: " << aLine;
	fseek(fInput, -1, SEEK_CUR);
	// Third line: size
	fscanf(fInput, "%d", &nWidth);
	std::cout << "\tWidth: " << nWidth;
	fscanf(fInput, "%d", &nHeight);
	std::cout << " Height: " << nHeight << std::endl;
	// Fourth line: max value
	fscanf(fInput, "%d", &nMaxGray);
	std::cout << "\tMax value: " << nMaxGray << std::endl;
	while (getc(fInput) != '\n');
	// Following lines: data
	thrust::host_vector<Npp8u> pSrc_Host(nWidth * nHeight);
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
		{
			pSrc_Host.push_back(fgetc(fInput));
			if (i < 5 && j < 5)
			{
				std::cout << fgetc(fInput) << std::endl;
			}
		}

	for (thrust::host_vector<Npp8u>::iterator i = pSrc_Host.begin(); i != pSrc_Host.begin()+25 ; i++)
	{
		std::cout << "pSrc_Host[" << &i << "] = " << *i << std::endl;
	}
	fclose(fInput);
	getchar();
	return pSrc_Host;
}

// Write PGM image.
void
WritePGM(char * sFileName, Npp8u * pDst_Host, int nWidth, int nHeight, int nMaxGray)
{
	FILE * fOutput = fopen(sFileName, "w+");
	if (fOutput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	char * aComment = "# Created by NPP";
	fprintf(fOutput, "P5\n%s\n%d %d\n%d\n", aComment, nWidth, nHeight, nMaxGray);
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			fputc(pDst_Host[i*nWidth + j], fOutput);
	fclose(fOutput);
}