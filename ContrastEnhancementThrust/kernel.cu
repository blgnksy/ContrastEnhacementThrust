#include "hip/hip_runtime.h"

#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <npp.h>
#include <stdio.h>
#include <windows.h>

struct muldiv_functor
{
	unsigned int a;

	muldiv_functor(unsigned int nConstant, unsigned int nNormalizer) {
		a = round(nConstant / nNormalizer); 
	}

	__host__ __device__
		Npp8u operator()(const Npp8u& x) const
	{
		return a*x ;
	}
};

double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		std::cout << "QueryPerformanceFrequency failed!\n";

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter()
{
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}

// Function Protypes.
thrust::host_vector<Npp8u>
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray);

void
WritePGM(char * sFileName, thrust::host_vector<Npp8u> pDst_Host, int nWidth, int nHeight, int nMaxGray);

int main()
{
	thrust::host_vector<Npp8u> pSrc_Host;
	int   nWidth, nHeight, nMaxGray;
	unsigned int nNormalizer;
	std::cout << "THRUST VERSION" << std::endl;

	// Load image to the host.
	std::cout << "Load PGM file." << std::endl;
	pSrc_Host = LoadPGM("lena_before.pgm", nWidth, nHeight, nMaxGray);
	StartCounter();
	thrust::device_vector<Npp8u> pDst_Dev = pSrc_Host;
	std::cout << "Host to Device Memory Copy Duration : " <<GetCounter() << " seconds." << std::endl;
	
	//Finding Minimum
	StartCounter();
	int nMin = thrust::reduce(pDst_Dev.begin(), pDst_Dev.end(),257, thrust::minimum<int>());
	std::cout << "Finding Minimum Execution Time : " << GetCounter() << " seconds." << std::endl;

	//Finding Maximum
	StartCounter();
	int nMax = thrust::reduce(pDst_Dev.begin(), pDst_Dev.end(), 0, thrust::maximum<int>());
	std::cout << "Finding Maximum Execution Time : " << GetCounter() << " seconds." << std::endl;
	printf("The minimum value is %d, and the maximum value is %d.\n", nMin, nMax);

	std::cout << "Subracting the minimum value." << std::endl;
	StartCounter();
	thrust::for_each(pDst_Dev.begin(), pDst_Dev.end(), thrust::placeholders::_1 -= nMin);
	std::cout << "Subraction Execution Time : " << GetCounter() << " seconds." << std::endl;
	/*for (thrust::device_vector<Npp8u>::iterator  i = pDst_Dev.begin(); i!= pDst_Dev.end(); i++)
	{
		*i -= nMin;
	}*/
	std::cout << "Subraction finished." << std::endl;

	// Compute the optimal nConstant and nScaleFactor for integer operation see GTC 2013 Lab NPP.pptx for explanation
	// I will prefer integer arithmetic, Instead of using 255.0f / (nMax_Host - nMin_Host) directly
	int nScaleFactor = 0;
	int nPower = 1;
	while (nPower * 255.0f / (nMax - nMin) < 255.0f)
	{
		nScaleFactor++;
		nPower *= 2;
	}
	unsigned int nConstant = 255.0f / (nMax - nMin) * (nPower / 2);
	
	nNormalizer = pow(2, (nScaleFactor - 1));
	
	std::cout << "Multiplying by the constant, and dividing by normalizer." << std::endl;
	StartCounter();
	thrust::transform(pDst_Dev.begin(), pDst_Dev.end(), pDst_Dev.begin(), muldiv_functor(nConstant, nNormalizer));
	std::cout << "Multiplication Execution Time : " << GetCounter() << " seconds." << std::endl;
	/*for (thrust::device_vector<Npp8u>::iterator i = pDst_Dev.begin(); i != pDst_Dev.end(); i++)
	{
		*i = static_cast<Npp8u>(*i * (nConstant/nNormalizer));
	}*/
	std::cout << "Multiplication, and division finished." << std::endl;
	// Output the result image.
	StartCounter();
	thrust::host_vector<Npp8u> pDst_Host=pDst_Dev;
	std::cout << "Device to Host Copy Duration : " << GetCounter() << " seconds." << std::endl;

	std::cout << "Output the PGM file." << std::endl;
	WritePGM("lena_after_GPU_Thrust.pgm", pDst_Host, nWidth, nHeight, nMaxGray);
	getchar();
    return 0;
}

// Load PGM file.
thrust::host_vector<Npp8u>
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray)
{
	char aLine[256];
	FILE * fInput = fopen(sFileName, "r");
	if (fInput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	// First line: version
	fgets(aLine, 256, fInput);
	std::cout << "\tVersion: " << aLine;
	// Second line: comment
	fgets(aLine, 256, fInput);
	std::cout << "\tComment: " << aLine;
	fseek(fInput, -1, SEEK_CUR);
	// Third line: size
	fscanf(fInput, "%d", &nWidth);
	std::cout << "\tWidth: " << nWidth;
	fscanf(fInput, "%d", &nHeight);
	std::cout << " Height: " << nHeight << std::endl;
	// Fourth line: max value
	fscanf(fInput, "%d", &nMaxGray);
	std::cout << "\tMax value: " << nMaxGray << std::endl;
	while (getc(fInput) != '\n');
	// Following lines: data
	thrust::host_vector<Npp8u> pSrc_Host(0);
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
		{
			pSrc_Host.push_back(fgetc(fInput));
		}
	fclose(fInput);
	return pSrc_Host;
}

// Write PGM image.
void
WritePGM(char * sFileName, thrust::host_vector<Npp8u> pDst_Host, int nWidth, int nHeight, int nMaxGray)
{
	FILE * fOutput = fopen(sFileName, "wb");
	if (fOutput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	char * aComment = "# Created by NPP";
	fprintf(fOutput, "P5\n%s\n%d %d\n%d\n", aComment, nWidth, nHeight, nMaxGray);
	for (thrust::host_vector<Npp8u>::iterator i = pDst_Host.begin(); i != pDst_Host.end(); i++)
	{
		fputc(*i, fOutput);
	}
			
	fclose(fOutput);
}